/*

Copyright 2016 Thomas Luu

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

*/

/*

File: ncx2cdfinv.cu

Computation of the noncentral chi-squared quantile function.

Based on:

Luu, T; (2016) Fast and accurate parallel computation of quantile functions for 
random number generation. Doctoral thesis, UCL (University College London).

http://discovery.ucl.ac.uk/1482128/

*/

#ifndef NCX2CDFINV
#define NCX2CDFINV

#include <hip/hip_math_constants.h>

#if 1
#define TOL 0.1
#else
#define TOL 0.01
#endif

__host__ __device__ inline double sankaran(double u, double k, double l)
{
  double h = 1.0 - HIP_TWOTHIRD * (k + l) * (k + l + l + l) / ((k + l + l) * (k + l + l));
  double p = (k + l + l) / ((k + l) * (k + l));
  double m = (h - 1.0) * (1.0 - (h + h + h));
  double mu = 1.0 + h * p * (h - 1.0 - (1.0 - h * 0.5) * m * p);
  double s = h * sqrt(p + p) * (1 + m * p * 0.5);
  double z = normcdfinv(u);
  double x = z * s + mu;
  return (k + l) * pow(x, 1.0 / h);
}

__host__ __device__ inline double v(double u, double k, double l, double c)
{
  return pow(c * u, 2.0 / k);
}

__host__ __device__ inline double v_inv(double x, double k, double l, double c)
{
  return pow(x, 0.5 * k) / c;
}

__host__ __device__ inline double luu(double u, double k, double l, double *u_split = 0)
{
  double c = 0.5 * exp2(0.5 * k) * exp(0.5 * l) * k * tgamma(0.5 * k);
  double vv = v(u, k, l, c);

  double v_split;
  if (u_split != 0) {
    double rg2 = (k * (2.0 + k)) / (k - l);
    double rg3 = (2.0 * k * k * (2.0 + k) * (2.0 + k) * (4.0 + k)) / (6.0 * l * l * (k - 1) + 2.0 * l * (8.0 - 5.0 * k) * k + k * k * (5.0 * k - 8.0));
    if (k == l) {
      v_split = cbrt(TOL * fabs(rg3));
    } else {
      v_split = sqrt(TOL * fabs(rg2));
    }
    *u_split = v_inv(v_split, k, l, c);
  }

  return vv;
}

__host__ __device__ inline double ncx2cdfinv(double u, double k, double l)
{
  if (u == 0.0) {
    return 0.0;
  }

  if (u == 1.0) {
#ifdef __CUDA_ARCH__
    return CUDART_INF;
#else
    return INFINITY;
#endif
  }

  double sankaran_approx = sankaran(u, k, l);

  double u_split;
  double luu_approx = luu(u, k, l, &u_split);

  if (isnan(sankaran_approx)) return luu_approx;
  return u < u_split ? luu_approx : sankaran_approx;
}

#endif

